#include "hip/hip_runtime.h"
#include "rtlflow.h"
#define TO_STR(x) #x
#define HEADER_FILE_NAME(Design) TO_STR(Design.h)

#include HEADER_FILE_NAME(Design)

#include <chrono>
#include <iostream>

RF::RTLflow rtlflow(GPU_THREADS);
RF::RTLflow &RF::Design::_rtlflow = rtlflow;

#define GET(x) *(rtlflow.get(x, 0))

int main(int argc, char **argv) {
  auto Dut = std::make_unique<RF::Design>();
  auto Cycles = std::atoi(argv[1]);

  auto Clock = [&]() {
    // Dut->clock = false;
    GET(Dut.get()->clock) = false;
    Dut->eval();

    GET(Dut.get()->clock) = true;
    Dut->eval();
  };

  //===--------------------------------------------------------------------===//
  // Model initialization and reset
  //===--------------------------------------------------------------------===//

  GET(Dut.get()->reset) = false;

  for (int i = 0; i < 1000; ++i) {
    GET(Dut.get()->reset) = i < 100;
    Clock();
  }

  //===--------------------------------------------------------------------===//
  // Simulation loop
  //===--------------------------------------------------------------------===//

  auto Start = std::chrono::system_clock::now();

  for (int i = 0; i < Cycles; ++i)
    Clock();

  auto End = std::chrono::system_clock::now();

  std::cout << std::chrono::duration_cast<std::chrono::microseconds>(End -
                                                                     Start)
                   .count()
            << std::endl;

  return 0;
}
